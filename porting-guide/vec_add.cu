#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

using namespace std;

#define ERRCHECK {                                                            \
    hipError_t err;                                                       \
    if ((err = hipGetLastError()) != hipSuccess) {                       \
        std::cout << "CUDA error: " << hipGetErrorString(err) << " : "    \
                  << __FILE__ << ", line " << __LINE__ << std::endl;       \
        exit(1);                                                           \
    }                                                                      \
}

#define TIMERSTART(label)                                                  \
    hipSetDevice(0);                                                      \
    hipEvent_t start##label, stop##label;                                 \
    float time##label;                                                     \
    hipEventCreate(&start##label);                                        \
    hipEventCreate(&stop##label);                                         \
    hipEventRecord(start##label, 0);

#define TIMERSTOP(label)                                                   \
        hipSetDevice(0);                                                  \
        hipEventRecord(stop##label, 0);                                   \
        hipEventSynchronize(stop##label);                                 \
        hipEventElapsedTime(&time##label, start##label, stop##label);     \
        std::cout << "TIMING: " << time##label << " ms (" << #label << ")" \
                  << std::endl;



__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceC) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceC[index] = deviceA[index] + deviceB[index];
}

__global__ void emptyKernel(float* deviceA, float* deviceB, float* deviceC) {}


void vectorAddCuda(int n, float* hostA, float* hostB, float* hostC) {
    int threadBlockSize = 512;

    /* Allocate the vectors on the GPU. */
    float *deviceA, *deviceB, *deviceC;

    hipMalloc((void **) &deviceA, n * sizeof(float));                                      ERRCHECK
    hipMalloc((void **) &deviceB, n * sizeof(float));                                      ERRCHECK
    hipMalloc((void **) &deviceC, n * sizeof(float));                                      ERRCHECK

    /* Copy the original vectors to the GPU. */
    TIMERSTART(host_to_device)
    hipMemcpy(deviceA, hostA, n*sizeof(float), hipMemcpyHostToDevice);                    ERRCHECK
    hipMemcpy(deviceB, hostB, n*sizeof(float), hipMemcpyHostToDevice);                    ERRCHECK
    TIMERSTOP(host_to_device)

    hipLaunchKernelGGL((emptyKernel), dim3(1), dim3(1), 0, 0, deviceA, deviceB, deviceC);                                       ERRCHECK

    /* Execute and time the kernel */
    TIMERSTART(kernel)
    hipLaunchKernelGGL((vectorAddKernel), dim3(n/threadBlockSize), dim3(threadBlockSize), 0, 0, deviceA, deviceB, deviceC);     ERRCHECK
    TIMERSTOP(kernel)

    TIMERSTART(device_to_host)
    /* Copy back results */
    hipMemcpy(hostC, deviceC, n * sizeof(float), hipMemcpyDeviceToHost);                  ERRCHECK
    TIMERSTOP(device_to_host)

    hipFree(deviceA);                                                                      ERRCHECK
    hipFree(deviceB);                                                                      ERRCHECK
    hipFree(deviceC);                                                                      ERRCHECK
}


int main(int argc, char* argv[]) {
    int n = 65536000;
    float* hostA = new float[n];
    float* hostB = new float[n];
    float* hostC = new float[n];

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device Number: %d\n", 0);
    printf("  Device name: %s\n\n", prop.name);

    /* initialize the vectors. */
    for(int i = 0; i < n; i++) {
        hostA[i] = i;
        hostB[i] = i;
    }

    vectorAddCuda(n, hostA, hostB, hostC);

    /* verify the resuls. */
    for(int i=0; i<n; i++) {
        if(hostC[i] != 2*i) {
            cout << "error in results! Element " << i << " is " << hostC[i] << ", but should be " << (2*i) << endl;
            exit(0);
        }
    }

    cout << "results OK!" << endl;

    delete[] hostA;
    delete[] hostB;
    delete[] hostC;
}
