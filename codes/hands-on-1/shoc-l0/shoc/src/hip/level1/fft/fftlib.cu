#include "hip/hip_runtime.h"
#include "cudacommon.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
/*#include <hipfft/hipfft.h>*/
#include "OptionParser.h"
#include "fftlib.h"

int fftDevice = -1;

bool do_dp;

//#define USE_CUFFT

#ifdef USE_CUFFT
hipfftHandle plan;
// Arrange blocks into 2D grid that fits into the GPU (for powers of two only)
inline dim3 grid2D(const int nblocks)
{
    int slices = 1;
    while (nblocks/slices > 65535)
    {
        slices *= 2;
    }
    return dim3(nblocks/slices, slices);
}

void printCUFFTError(const hipfftResult res)
{
    if (res != HIPFFT_SUCCESS)
    {
        cout << "CUFFT Error: ";
        if (res == HIPFFT_INVALID_PLAN)
        {
            cout << "Invalid Plan.\n";
        }
        else if (res == HIPFFT_INVALID_VALUE)
        {
            cout << "Invalid Value.\n";
        }
        else if (res == HIPFFT_INTERNAL_ERROR)
        {
            cout << "Internal Error .\n";
        }
        else if (res == HIPFFT_EXEC_FAILED)
        {
            cout << "FFT Exec failed.\n";
        }
        else if (res == HIPFFT_SETUP_FAILED)
        {
            cout << "Setup failed.\n";
        }
        else if (res == HIPFFT_UNALIGNED_DATA)
        {
            cout << "Unaligned data (unused).\n";
        }
    }
}

#else
#include "codelets.h"
#endif

template <class T2> __global__ void
chk512_device(const T2* __restrict__ work, const int half_n_cmplx,
    char* __restrict__ fail)
{
    int i, tid = threadIdx.x;
    T2 a[8], b[8];

    work += (blockIdx.y * gridDim.x + blockIdx.x) * 512 + tid;

    for (i = 0; i < 8; i++)
    {
        a[i] = work[i*64];
    }

    for (i = 0; i < 8; i++)
    {
        b[i] = work[half_n_cmplx+i*64];
    }

    for (i = 0; i < 8; i++)
    {
        if (a[i].x != b[i].x || a[i].y != b[i].y)
        {
            *fail = 1;
        }
    }
}


template <class T2> __global__ void
norm512_device(T2* __restrict__ work)
{
    int i, tid = threadIdx.x;

    work += (blockIdx.y * gridDim.x + blockIdx.x) * 512 + tid;

    for (i = 0; i < 8; i++)
    {
        work[i*64].x /= 512;
        work[i*64].y /= 512;
    }
}


void
init(OptionParser& op, const bool _do_dp, const int n_ffts)
{
    do_dp = _do_dp;
    if (fftDevice == -1)
    {
        if (op.getOptionVecInt("device").size() > 0)
        {
            fftDevice = op.getOptionVecInt("device")[0];
        }
        else
        {
            fftDevice = 0;
        }
        hipSetDevice(fftDevice);
        hipGetDevice(&fftDevice);
    }
#ifdef USE_CUFFT
    hipfftResult res;
    cerr << "init: initing plan, n_ffts=" << n_ffts << endl;
    if (do_dp)
    {
        res = hipfftPlan1d(&plan, 512, HIPFFT_Z2Z, n_ffts);
    }
    else
    {
        res = hipfftPlan1d(&plan, 512, HIPFFT_C2C, n_ffts);
    }
    if (res != HIPFFT_SUCCESS)
    {
        cout << "CUFFT Error in plan.\n";
    }
    else
    {
        cerr <<  "success...\n";
    }
#endif

}


void
forward(void* work, const int n_ffts)
{
#ifdef USE_CUFFT
    hipfftResult res;
    if (do_dp)
    {
        res = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)work,
            (hipfftDoubleComplex*)work, HIPFFT_FORWARD);
    }
    else
    {
        res = hipfftExecC2C(plan, (hipfftComplex*)work,
            (hipfftComplex*)work, HIPFFT_FORWARD);
    }
    printCUFFTError(res);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#else
    if (do_dp)
    {
        hipLaunchKernelGGL((FFT512_device<double2, double>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL((FFT512_device<float2, float>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#endif
}


void
inverse(void* work, const int n_ffts)
{
#ifdef USE_CUFFT
    hipfftResult res;
    if (do_dp)
    {
        res = hipfftExecZ2Z(plan, (hipfftDoubleComplex*)work,
            (hipfftDoubleComplex*)work, HIPFFT_BACKWARD);
    }
    else
    {
        res = hipfftExecC2C(plan, (hipfftComplex*)work,
            (hipfftComplex*)work, HIPFFT_BACKWARD);
    }
    printCUFFTError(res);

    // normalize data...
    if (do_dp)
    {
        hipLaunchKernelGGL((norm512_device<double2>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL((norm512_device<float2>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
#else
    if (do_dp)
    {
        hipLaunchKernelGGL((IFFT512_device<double2, double>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (double2*)work);
    }
    else
    {
        hipLaunchKernelGGL((IFFT512_device<float2, float>), dim3(grid2D(n_ffts)), dim3(64), 0, 0, (float2*)work);
    }
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
    // normalization built in to inverse...
#endif
}


int
check(void* work, void* check, const int half_n_ffts, const int half_n_cmplx)
{
    char result;

    if (do_dp)
    {
        hipLaunchKernelGGL((chk512_device<double2>), dim3(grid2D(half_n_ffts)), dim3(64), 0, 0, 
            (double2*)work, half_n_cmplx, (char*)check);
    }
    else
    {
        hipLaunchKernelGGL((chk512_device<float2>), dim3(grid2D(half_n_ffts)), dim3(64), 0, 0, 
            (float2*)work, half_n_cmplx, (char*)check);
    }
    hipMemcpy(&result, check, 1, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR();

    return result;
}


void
allocHostBuffer(void** bufferp, unsigned long bytes)
{
    hipHostMalloc(bufferp, bytes);
    CHECK_CUDA_ERROR();
}

void
allocDeviceBuffer(void** bufferp, unsigned long bytes)
{
    hipMalloc(bufferp, bytes);
    CHECK_CUDA_ERROR();
}

void
freeHostBuffer(void* buffer)
{
    hipHostFree(buffer);
    CHECK_CUDA_ERROR();
}


void
freeDeviceBuffer(void* buffer)
{
    hipFree(buffer);
}

void
copyToDevice(void* to_device, const void* from_host,
    const unsigned long bytes)
{
    hipMemcpy(to_device, from_host, bytes, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();
}

void
copyFromDevice(void* to_host, const void* from_device,
    const unsigned long bytes)
{
    hipMemcpy(to_host, from_device, bytes, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR();
}

